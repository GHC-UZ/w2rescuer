#include <iostream>
#include <hip/hip_runtime.h>

__global__ void hello() {
    printf("\n************");
    printf("\nHi from GPU!");
    printf("\n************\n\n");
}

int main() {

    // Step 1: Get the number of GPUs available
    int numGPUs = 0;
    hipGetDeviceCount(&numGPUs);
    std::cout << "Number of GPUs available: " << numGPUs << std::endl;

    if (numGPUs == 0) {
        std::cerr << "No GPUs found!" << std::endl;
        return -1;
    }



    // Step 2: List properties of each GPU
    for (int i = 0; i < numGPUs; i++) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);

        std::cout << "GPU " << i << ": " << props.name << std::endl;
    } 



    // Step 3: Select a GPU (e.g., GPU 0)
    int selectedGPU = 0;
    hipSetDevice(selectedGPU);  




    // Step 4: Verify the selected GPU
    hipDeviceProp_t selectedProps;
    hipGetDeviceProperties(&selectedProps, selectedGPU);

    std::cout << "\nSelected GPU: " << selectedProps.name << std::endl; 
    std::cout << "  Compute Capability: " << selectedProps.major << "." << selectedProps.minor << std::endl; 
    std::cout << "  Multiprocessors (SM): " << selectedProps.multiProcessorCount << std::endl;
    std::cout << "  Maximun threads per block: " << selectedProps.maxThreadsPerBlock << std::endl;
    std::cout << "  Maximun number of blocks (in x): " << selectedProps.maxGridSize[0] << std::endl;
    std::cout << "  Maximun threads in each SM: " << selectedProps.maxThreadsPerMultiProcessor << std::endl;
    



    // Step 5: Get total and free memory
    size_t freeMemory, totalMemory;
    hipMemGetInfo(&freeMemory, &totalMemory);

    std::cout << "  Total Global Memory: " << totalMemory / (1024 * 1024) << " MB" << std::endl;
    std::cout << "  Free Memory: " << freeMemory / (1024 * 1024) << " MB" << std::endl;



    // Launch kernel
    hello <<<1,1>>> ();

    // Check for launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Error launching kernel: " << hipGetErrorString(err) << std::endl;
        return -1;
    }




    // Synchronize and check for execution errors
    hipError_t sync_err = hipDeviceSynchronize();
    if (sync_err != hipSuccess) {
        std::cerr << "Error synchronizing: " << hipGetErrorString(sync_err) << std::endl;
        return -1;
    }




    // Flush stdout to ensure printf output is displayed
    fflush(stdout);

    return 0;
}
