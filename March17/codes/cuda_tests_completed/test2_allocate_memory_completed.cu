#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>

// Kernel functions for memory allocation
__global__ void checkMemoryAllocate(int N, size_t size, double *d_A) {

    int ithread = threadIdx.x;
    int iblock = blockIdx.x; 
    int idx = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    if (idx < N) {

        printf("<<gpu>> Thread %d in block %d - d_A[%d] = %.0lf \n", ithread, iblock, idx, d_A[idx]);

    }
       
}


__global__ void registerMemoryAllocate(int N, size_t size, double *d_A) {

    int ithread = threadIdx.x;
    int iblock = blockIdx.x; 
    int idx = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    //Declare registers
    double rA;

    if (idx < N) {

        // Use the register memory
        rA = (double)(idx);

        // Write the result back to global memory
        d_A[idx] = rA;

        printf("<<gpu>> Thread %d in block %d - d_A[%d] = %.0lf \n", ithread, iblock, idx, d_A[idx]);

    }
       
}


__global__ void sharedMemoryAllocate(int N, size_t size, double *d_A) {

    int ithread = threadIdx.x;
    int iblock = blockIdx.x; 
    int idx = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    // Declare dynamic shared memory
    extern __shared__ double shared_A[];

    // Allocate shared memory
    if (idx < N) {
        shared_A[ithread] = (double)(idx);
    }

    // Synchronize to ensure all threads have written to shared memory
    __syncthreads();        

    if (idx < N) {
        // Write the result back to global memory
        d_A[idx] = shared_A[ithread];

        printf("<<gpu>> Thread %d in block %d - d_A[%d] = %.0lf \n", ithread, iblock, idx, d_A[idx]);

    }
    
}
                                       


int main() {

    hipError_t err; // CUDA error
    double exec_time=0.0; //timer 
    clock_t stime1, stime2;

    // Size of the vector
    int N = 1000;

    // Allocate memory on the host
    size_t size = N*sizeof(double);
    double *h_A = (double*) malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = (double)i;
    }



    // Select a GPU
    int selectedGPU = 0; // Change this to select your asigned GPU
    hipSetDevice(selectedGPU);  

    // Get the device propoerties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, selectedGPU);    

    // Get total and free memory
    size_t freeMemory, totalMemory;
    hipMemGetInfo(&freeMemory, &totalMemory);
    std::cout << "  Total Global Memory: " << totalMemory / (1024 * 1024) << " MB" << std::endl;
    std::cout << "  Free Memory: " << freeMemory / (1024 * 1024) << " MB" << std::endl;
           
    getchar(); //Pause point




    // Allocate memory on the device
    double *d_A;
    err = hipMalloc((void**) &d_A, size);

    // Check if the allocation was successful
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        return 1;
    }    

    // Check the updated free memory
    hipMemGetInfo(&freeMemory, &totalMemory);
    std::cout << "  Updated free Memory: " << freeMemory / (1024 * 1024) << " MB" << std::endl;

    getchar(); //Pause point





    //Start IO time .....................................
    stime1=clock();

    // Launch free-load kernel
    // int threadsPerBlock = 256;
    // int blocksPerGrid = N/threadsPerBlock+1;

    // checkMemoryAllocate <<<blocksPerGrid, threadsPerBlock>>> (N, size, d_A);




    //Launch register allocation kernel
    // int threadsPerBlock = 256;
    // int blocksPerGrid = N/threadsPerBlock+1;

    // registerMemoryAllocate <<<blocksPerGrid, threadsPerBlock>>> (N, size, d_A);



    //Launch shared allocation kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = N/threadsPerBlock+1;    
    size_t requiredSharedMemory = threadsPerBlock*sizeof(double);
    std::cout << "  Maximum shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl; 
    std::cout << "  Required shared memory per block: " << requiredSharedMemory << " bytes" << std::endl;
    
    sharedMemoryAllocate <<<blocksPerGrid, threadsPerBlock, requiredSharedMemory>>> (N, size, d_A);



    // Synchronize device
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Error en cudaDeviceSynchronize: " << hipGetErrorString(err) << std::endl;
    }

    stime2=clock();
    exec_time += double(stime2-stime1)/CLOCKS_PER_SEC;    
    std::cerr << "  Execution time: " << exec_time << std::endl; 
    //End IO time .....................................   

    getchar(); //Pause point





    // Free device memory
    hipFree(d_A);

    // Check the updated free memory
    hipDeviceSynchronize();
    hipMemGetInfo(&freeMemory, &totalMemory);
    std::cout << "  Final free Memory: " << freeMemory / (1024 * 1024) << " MB" << std::endl;

    // Free host memory
    free(h_A);

    return 0;
}

