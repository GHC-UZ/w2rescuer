#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to add elements of two arrays
__global__ void vectorAdd(int N, double *A, double *B, double *C) {
    
    int ithread = threadIdx.x;
    int iblock = blockIdx.x; 
    int idx = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    if (idx < N) {     
        C[idx] = A[idx] + B[idx];
    
        printf("<<gpu>> Thread %d in block %d - d_C[%d] = %.0lf \n", ithread, iblock, idx, C[idx]);
    
    }

}


__device__ void addDoubleData(double *A, double *B, double *C) {
     
    (*C) = (*A) + (*B);

}


__global__ void deviceVectorAdd(int N, double *A, double *B, double *C) {
    
    int ithread = threadIdx.x;
    int iblock = blockIdx.x; 
    int idx = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    if (idx < N) { 
        addDoubleData( &(A[idx]), &(B[idx]), &(C[idx]) );    
        
        printf("<<gpu>> Thread %d in block %d - d_C[%d] = %.0lf \n", ithread, iblock, idx, C[idx]);  
    }

}



__global__ void coalescentVectorAdd(int N, double *A, double *B, double *C) {
    
    int ithread = threadIdx.x;
    int iblock = blockIdx.x; 
    int idx = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    if (idx < N) {     

        for(int j=0; j<100000;j++){
            C[idx] = A[idx] + B[idx];
        }           

        //printf("<<gpu>> Thread %d in block %d - d_C[%d] = %.0lf \n", ithread, iblock, idx, C[idx]);
    
    }

}


__global__ void nonCoalescentVectorAdd(int N, double *A, double *B, double *C, int stride) {
    
    int ithread = threadIdx.x;
    int iblock = blockIdx.x; 
    int idx = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    if (idx < N) {
        for(int j=0; j<100000;j++){
            C[idx] = A[idx*stride] + B[idx*stride];
        }

        //printf("<<gpu>> Thread %d in block %d - d_C[%d] = %.0lf \n", ithread, iblock, idx, C[idx]);
    
    }

}





int main() {

    hipError_t err; // CUDA error
    double exec_time=0.0; //timer 
    clock_t stime1, stime2;
    int threadsPerBlock;
    int blocksPerGrid;



    // Size of the vectors
    int N = 1000; // 1 million elements

    // Allocate memory on the host
    size_t size = N*sizeof(double);
    double *h_A = (double*) malloc(size);
    double *h_B = (double*) malloc(size);
    double *h_C = (double*) malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = (double)i;
        h_B[i] = 0.5;
    }



    // Non-coalescent array
    const int stride = 8; //Separation between memory positions
    double *h_A_nc = (double*) malloc(size*stride);
    double *h_B_nc = (double*) malloc(size*stride);

    // Initialize host vectors
    for (int i = 0; i < N*stride; i++) {
        h_A_nc[i] = (double)(i/stride);
        
        //std::cout << "  h_A_nc[" << i << "] = " << h_A_nc[i] << std::endl;

        h_B_nc[i] = 0.5;
    }    

  



   // Select a GPU
    int selectedGPU = 0; // Change this to select your asigned GPU
    hipSetDevice(selectedGPU);  

    // Get the device propoerties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, selectedGPU);    

    // Get total and free memory
    size_t freeMemory, totalMemory;
    hipMemGetInfo(&freeMemory, &totalMemory);

    std::cout << "  Total Global Memory: " << totalMemory / (1024 * 1024) << " MB" << std::endl;
    std::cout << "  Free Memory: " << freeMemory / (1024 * 1024) << " MB" << std::endl;





    // Allocate memory on the device
    double *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, size);
    hipMalloc((void**) &d_B, size);
    hipMalloc((void**) &d_C, size);

    // Non-coalescent arrays allocation on the device
    double *d_A_nc, *d_B_nc;
    hipMalloc((void**) &d_A_nc, size*stride);
    hipMalloc((void**) &d_B_nc, size*stride);

    hipMemGetInfo(&freeMemory, &totalMemory);
    std::cout << "  Updated free Memory: " << freeMemory / (1024 * 1024) << " MB" << std::endl;




    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Non-coalescent arrays copy from host to device
    hipMemcpy(d_A_nc, h_A_nc, size*stride, hipMemcpyHostToDevice);
    hipMemcpy(d_B_nc, h_B_nc, size*stride, hipMemcpyHostToDevice);    

    getchar(); //Pause point



    //Launch parallel kernel
    //Start IO time .....................................
    stime1=clock();

    //All threads in a single block
    // std::cout << "  Number of tasks: " << N << std::endl;
    // std::cout << "  Maximun threads per block: " << prop.maxThreadsPerBlock << std::endl;
    // getchar();

    // vectorAdd <<<1, N>>> (N, d_A, d_B, d_C);


    //Launch the optimized kernel
    // threadsPerBlock = 256;
    // blocksPerGrid = N/threadsPerBlock+1;

    // vectorAdd <<<blocksPerGrid, threadsPerBlock>>> (N, d_A, d_B, d_C);

    //Launch device funtion
    // threadsPerBlock = 256;
    // blocksPerGrid = N/threadsPerBlock+1;

    // deviceVectorAdd <<<blocksPerGrid, threadsPerBlock>>> (N, d_A, d_B, d_C); 

    //Launch the coalescent kernel
    threadsPerBlock = 256;
    blocksPerGrid = N/threadsPerBlock+1;

    coalescentVectorAdd <<<blocksPerGrid, threadsPerBlock>>> (N, d_A, d_B, d_C);


    // Sincronizar y comprobar errores
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Error en cudaDeviceSynchronize: " << hipGetErrorString(err) << std::endl;
    }

    stime2=clock();
    exec_time += double(stime2-stime1)/CLOCKS_PER_SEC;    
    std::cerr << "  Execution time: " << exec_time << std::endl; 
    //End IO time .....................................  






    //Start IO time .....................................
    stime1=clock();

    //Launch non-coalescent access
    threadsPerBlock = 256;
    blocksPerGrid = N/threadsPerBlock+1;

    nonCoalescentVectorAdd <<<blocksPerGrid, threadsPerBlock>>> (N, d_A_nc, d_B_nc, d_C, stride);

    // Sincronizar y comprobar errores
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Error en cudaDeviceSynchronize: " << hipGetErrorString(err) << std::endl;
    }

    stime2=clock();
    exec_time += double(stime2-stime1)/CLOCKS_PER_SEC;    
    std::cerr << "  Non-coalescent execution time: " << exec_time << std::endl; 
    //End IO time .....................................   

    getchar(); //Pause point





    // Copy the result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    //Verify the result
    for (int i = 0; i < N; i++) {
        std::cout << "  h_C[" << i << "] = " << h_C[i] << std::endl;
    }




    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Check the updated free memory
    hipDeviceSynchronize();
    hipMemGetInfo(&freeMemory, &totalMemory);
    std::cout << "  Final free Memory: " << freeMemory / (1024 * 1024) << " MB" << std::endl;


    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
